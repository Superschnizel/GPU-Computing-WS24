#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <iostream>
#include <chrono>

#define GRIDSIZE 16.0

void check(hipError_t err, std::string msg) {
    if (err != hipSuccess) {
        std::cerr << msg << "(error code:" << hipGetErrorString(err) << ")";
        exit(EXIT_FAILURE);
    }
}

void init(int32_t size, int32_t *vec_a, int32_t *vec_b, int32_t *mat) {
    // std::random_device dev;
    std::mt19937 prng(2024);
    std::uniform_int_distribution <int32_t> distrib(-16, 16);

    for (auto i = 0; i < size; i++) {
        vec_a[i] = distrib(prng);
        vec_b[i] = distrib(prng);
    }

    for (auto i = 0; i < size * size; i++)
        mat[i] = distrib(prng);
}

void compute(int32_t size, int32_t *vec_a, int32_t *vec_b, int32_t *mat, int32_t *out) {
    auto tmp = (int32_t *) malloc(sizeof(int32_t) * size);
    for (auto i = 0; i < size; i++)
        tmp[i] = vec_a[i] + vec_b[i];

    for (auto i = 0; i < size; i++) {
        out[i] = 0;
        for (auto j = 0; j < size; j++)
            out[i] += tmp[j] * mat[i * size + j];
    }
    free(tmp);
}

__global__ void vectorAdd(const int32_t *A, const int32_t *B,
                          int32_t *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

__global__ void setZero(int32_t *A) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    A[i] = 0;
}

__global__ void matrixMult(const int32_t size, const int32_t *V, const int32_t *M, int32_t *out) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    out[i] += V[j] * M[i * size + j];
}

void pretty_print(int32_t size, int32_t *vec_a, int32_t *vec_b, int32_t *mat) {
    std::cout << "Vec A:" << std::endl;
    for (auto i = 0; i < size; i++)
        std::cout << vec_a[i] << std::endl;

    std::cout << "Vec B:" << std::endl;
    for (auto i = 0; i < size; i++)
        std::cout << vec_b[i] << std::endl;

    std::cout << "Matrix:" << std::endl;
    for (auto i = 0; i < size; i++) {
        for (auto j = 0; j < size; j++)
            std::cout << mat[i * size + j] << " ";

        std::cout << std::endl;
    }
}

int main() {
    // int32_t size = 3;
    int32_t size = 32768;

    hipError_t err = hipSuccess;

    auto h_vec_a = (int32_t *) malloc(sizeof(int32_t) * size);
    auto h_vec_b = (int32_t *) malloc(sizeof(int32_t) * size);
    // Flat Buffer for matrix
    auto h_mat = (int32_t *) malloc(sizeof(int32_t * ) * size * size);
    auto h_out = (int32_t *) malloc(sizeof(int32_t) * size);

    init(size, h_vec_a, h_vec_b, h_mat);

    int32_t *d_vec_a = NULL;
    err = hipMalloc((void **) &d_vec_a, size);
    check(err, "Failed to allocate device vector A");

    int32_t *d_vec_b = NULL;
    err = hipMalloc((void **) &d_vec_b, size);
    check(err, "Failed to allocate device vector B");

    int32_t *d_out = NULL;
    err = hipMalloc((void **) &d_out, size);
    check(err, "Failed to allocate device vector OUT");

    int32_t *d_mat = NULL;
    err = hipMalloc((void **) &d_mat, size * size);
    check(err, "Failed to allocate device Matrix");

    std::cout << "Copy input data from the host memory to the CUDA device\n";
    err = hipMemcpy(d_vec_a, h_vec_a, size, hipMemcpyHostToDevice);
    check(err, "Failed to copy vector A from host to device");

    err = hipMemcpy(d_vec_b, h_vec_b, size, hipMemcpyHostToDevice);
    check(err, "Failed to copy vector B from host to device");

    err = hipMemcpy(d_mat, h_mat, size * size, hipMemcpyHostToDevice);
    check(err, "Failed to copy Matrix from host to device");
    // pretty_print(size, vec_a, vec_b, mat);


    int numberOfThreadsPerBlock = (int) GRIDSIZE * GRIDSIZE;
    int oneDimBlockCount = (int) ceil(size / (double) numberOfThreadsPerBlock);

    vectorAdd << oneDimBlockCount, numberOfThreadsPerBlock >> (d_vec_a, d_vec_b, d_out, size);

    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if (cudaerror != hipSuccess) {
        fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror)); // if error, output error
    }

    setZero << OneDimBlockCount, NumberOfThreadsPerBlock >> (d_vec_a);

    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if (cudaerror != hipSuccess) {
        fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror)); // if error, output error

    }

    dim3 threadsPerBlock(GRIDSIZE, GRIDSIZE, 1);
    const int32_t twoDimBlockCount = ceil(size / GRIDSIZE);
    dim3 numBlocks(twoDimBlockCount, twoDimBlockCount, 1);

    matrixMult << numBlocks, threadsPerBlock >> (size, d_out, d_mat, d_vec_a);

    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if (cudaerror != hipSuccess)
        fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror)); // if error, output error

    auto start = std::chrono::system_clock::now();
    auto end = std::chrono::system_clock::now();

    std::cout << "First 3 entries of Out Vec:" << std::endl;
    for (int32_t i = 0; i < 3; i++)
        std::cout << out[i] << std::endl;

    std::chrono::duration<double> elapsed_seconds = end - start;
    std::cout << "Elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

    free(vec_a);
    free(vec_b);
    free(mat);
    free(out);

    return 0;
}
