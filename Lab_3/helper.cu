
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <chrono>

#define CUDA_CALL(x)                                                                                          \
    do                                                                                                        \
    {                                                                                                         \
        hipError_t error = x;                                                                                \
        if (error != hipSuccess)                                                                             \
        {                                                                                                     \
            const char *cuda_err_str = hipGetErrorString(error);                                             \
            std::cerr << "Cuda Error at" << __FILE__ << ":" << __LINE__ << ": " << cuda_err_str << std::endl; \
            return EXIT_FAILURE;                                                                              \
        }                                                                                                     \
    } while (0)

#define CURAND_CALL(x)                                                                                    \
    do                                                                                                    \
    {                                                                                                     \
        hiprandStatus_t error = x;                                                                         \
        if (error != HIPRAND_STATUS_SUCCESS)                                                               \
        {                                                                                                 \
            std::cerr << "CudaRand Error " << error << " at" << __FILE__ << ":" << __LINE__ << std::endl; \
            return EXIT_FAILURE;                                                                          \
        }                                                                                                 \
    } while (0)

#define CHECK_ALLOC(x)                                                                 \
    do                                                                                 \
    {                                                                                  \
        if ((x) == NULL)                                                               \
        {                                                                              \
            std::cerr << "Alloc Error at" << __FILE__ << ":" << __LINE__ << std::endl; \
            return EXIT_FAILURE;                                                       \
        }                                                                              \
    } while (0)

__global__ void scale_vec(float *in_d, float x, size_t size)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        in_d[i] *= x;
    }
}

int random_init(size_t size, float *in_d, float *in_h)
{
    hiprandGenerator_t gen;
    // Create PRNG
    CURAND_CALL(hiprandCreateGenerator(&gen,
                                      HIPRAND_RNG_PSEUDO_DEFAULT));
    // Set seed
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,
                                                   2048ULL));

    // Generate size floats on device
    CURAND_CALL(hiprandGenerateUniform(gen, in_d, size));

    // Scale by two so it does not get boring
    scale_vec<<<ceil(size / 256.0), 256>>>(in_d, 2.0, size);

    // Copy device memory to host
    CUDA_CALL(hipMemcpy(in_h, in_d, size * sizeof(float),
                         hipMemcpyDeviceToHost));

    CURAND_CALL(hiprandDestroyGenerator(gen));

    return EXIT_SUCCESS;
}